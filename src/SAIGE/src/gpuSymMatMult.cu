#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "gpuSymMatMult.hpp"

#define IDX2C(i,j,ld)  (((j)*(ld))+(i))

gpuSymMatMult::gpuSymMatMult()
{
    m_isLoaded = false;
    m_rows = m_cols = 0;
    m_handle = nullptr;
    m_A = nullptr;
    m_x = nullptr;
    m_y = nullptr;
    m_z = nullptr;
}

gpuSymMatMult::~gpuSymMatMult()
{
    if (m_A) hipFree(m_A);
    if (m_x) hipFree(m_x);
    if (m_y) hipFree(m_y);
    if (m_z) hipFree(m_z);
    hipblasDestroy((hipblasHandle_t)m_handle);
}

int gpuSymMatMult::set_matrix(int rank, size_t g_col_start, size_t n_rows, size_t n_cols, const float *A)
{
    hipError_t cudaStat;
    hipblasStatus_t cublasStat;
    size_t gpu_free, gpu_total;
    int device_id = -1;
    int ret = EXIT_SUCCESS;

    m_global_col_start = g_col_start;
    m_rows = n_rows;
    m_cols = n_cols;
    std::cout << "[" << rank << "] "
              << "A in nvcc = " << A << std::endl;

/*
    cudaStat = hipSetDevice(rank);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipSetDevice() has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }
*/
    cudaStat = hipGetDevice(&device_id);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipGetDevice() has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }

    std::cout << "[" << rank << "] "
              << "GPU ID: " << device_id << std::endl;

    cudaStat = hipMemGetInfo(&gpu_free, &gpu_total);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMemGetInfo() has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }

    std::cout << "[" << rank << "] "
              << "GPU memory: " << gpu_free << " / " << gpu_total 
              << " requesting: " << m_rows*m_cols*sizeof(*m_A) << std::endl;

    cudaStat = hipMalloc((void**)&m_A, m_rows*m_cols*sizeof(*m_A));
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMalloc() for A has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }
    cudaStat = hipMalloc((void**)&m_x, m_rows*sizeof(*m_x));
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMalloc() for x has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }
    cudaStat = hipMalloc((void**)&m_y, m_cols*sizeof(*m_y));
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMalloc() for y has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }
    cudaStat = hipMalloc((void**)&m_z, m_rows*sizeof(*m_z));
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMalloc() for z has failed: " << cudaStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }
    cublasStat = hipblasCreate((hipblasHandle_t *)&m_handle);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipblasCreate() has failed: " << cublasStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }
    cublasStat = hipblasSetMatrix(m_rows, m_cols, sizeof(*m_A), A, m_rows, m_A, m_rows);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipblasSetMatrix() has failed: " << cublasStat << std::endl;
        ret = EXIT_FAILURE;
        goto out;
    }

    m_isLoaded = true;
out:
    if (ret == EXIT_FAILURE) {
        m_rows = m_cols = 0;
        m_global_col_start = 0;
        if (m_A) hipFree(m_A);
        if (m_x) hipFree(m_x);
        if (m_y) hipFree(m_y);
        if (m_z) hipFree(m_z);
        if (m_handle) hipblasDestroy((hipblasHandle_t)m_handle);

        m_isLoaded = false;
        m_A = nullptr;
        m_x = nullptr;
        m_y = nullptr;
        m_z = nullptr;
        m_handle = nullptr;
    }
    return ret;
}

// Perform BB^T*x in two steps where B is a submatrix from A formed by B = A_{:,start_col:(end_col-1)}.
//   i)  y = B^T*x
//   ii) z = By
int gpuSymMatMult::sym_sgemv_range(int rank, size_t g_start_col, size_t g_end_col, size_t n_elem, const float *x, float *ret)
{
    hipError_t cudaStat;
    hipblasStatus_t cublasStat;
    size_t start_col, end_col, n_subcols;
    const float *B;
    float alpha = 1.0, beta = 0.0;

    if (n_elem != m_rows) {
        std::cerr << "[" << rank << "] "
                  << "Error: size mismatch: "
                  << "m_rows: " << m_rows << " n_elem: " << n_elem << std::endl;
        return EXIT_FAILURE;
    }
    if (g_start_col >= g_end_col) {
        std::cerr << "[" << rank << "] "
                  << "Error: invalid range = ["
                  << g_start_col << ", " << g_end_col << ")" << std::endl;
        return EXIT_FAILURE;
    }

    memset(ret, 0, n_elem*sizeof(*ret));

    // if there is no overlap, return immediately.
    if (g_end_col <= m_global_col_start || g_start_col >= m_global_col_start+m_cols) {
        return EXIT_SUCCESS;
    }

    start_col = max(g_start_col, m_global_col_start) - m_global_col_start;
    end_col = min(g_end_col, m_global_col_start+m_cols) - m_global_col_start;

    cudaStat = hipMemcpy(m_x, x, n_elem*sizeof(*x), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMemcpy (host to Device) failed: " << cudaStat
                  << " m_x = " << m_x << " x = " << x << std::endl;
        return EXIT_FAILURE;
    }

    n_subcols = end_col - start_col;
    B = m_A + m_rows*start_col;

    cublasStat = hipblasSgemv((hipblasHandle_t)m_handle, HIPBLAS_OP_T, m_rows, n_subcols, &alpha, B, m_rows, m_x, 1, &beta, m_y, 1);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipblasSgemv failed (HIPBLAS_OP_T): " << cublasStat << std::endl;
        return EXIT_FAILURE;
    }
    cublasStat = hipblasSgemv((hipblasHandle_t)m_handle, HIPBLAS_OP_N, m_rows, n_subcols, &alpha, B, m_rows, m_y, 1, &beta, m_z, 1);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipblasSgemv failed (HIPBLAS_OP_N): " << cublasStat << std::endl;
        return EXIT_FAILURE;
    }
    cudaStat = hipMemcpy(ret, m_z, n_elem*sizeof(*ret), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMemcpy (device to host) failed: " << cudaStat << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;

}

// Perform AA^T*x in two steps.
//   i)  y = A^T*x
//   ii) z = Ay
int gpuSymMatMult::sym_sgemv(int rank, size_t n_elem, const float *x, float *ret)
{
    hipError_t cudaStat;
    hipblasStatus_t cublasStat;
    float alpha = 1.0, beta = 0.0;

    if (n_elem != m_rows) {
        std::cerr << "[" << rank << "] "
                  << "Error: size mismatch: "
                  << "m_rows: " << m_rows << " n_elem: " << n_elem << std::endl;
        return EXIT_FAILURE;
    }

    memset(ret, 0, n_elem*sizeof(*ret));
    cudaStat = hipMemcpy(m_x, x, n_elem*sizeof(*x), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMemcpy (host to Device) failed: " << cudaStat
                  << " m_x = " << m_x << " x = " << x << std::endl;
        return EXIT_FAILURE;
    }
    cublasStat = hipblasSgemv((hipblasHandle_t)m_handle, HIPBLAS_OP_T, m_rows, m_cols, &alpha, m_A, m_rows, m_x, 1, &beta, m_y, 1);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipblasSgemv failed (HIPBLAS_OP_T): " << cublasStat << std::endl;
        return EXIT_FAILURE;
    }
    cublasStat = hipblasSgemv((hipblasHandle_t)m_handle, HIPBLAS_OP_N, m_rows, m_cols, &alpha, m_A, m_rows, m_y, 1, &beta, m_z, 1);
    if (cublasStat != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipblasSgemv failed (HIPBLAS_OP_N): " << cublasStat << std::endl;
        return EXIT_FAILURE;
    }
    cudaStat = hipMemcpy(ret, m_z, n_elem*sizeof(*ret), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        std::cerr << "[" << rank << "] "
                  << "Error: hipMemcpy (device to host) failed: " << cudaStat << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
